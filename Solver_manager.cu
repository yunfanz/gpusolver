//#include "cuSolverDn_AtA.cu"
#include "Solver_manager.hh"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <cmath> 
#include <hip/hip_runtime.h>
#include "SI.h"
#include "hipblas.h"
#include "hipsparse.h"
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"
#include "helper_cusolver.h"

int linearSolverCHOL(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    int bufferSize = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    checkCudaErrors(hipsolverDnSpotrf_bufferSize(handle, uplo, n, (float*)Acopy, lda, &bufferSize));

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));


    // prepare a copy of A because potrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

    checkCudaErrors(hipsolverDnSpotrf(handle, uplo, n, A, lda, buffer, bufferSize, info));

    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: Cholesky factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipsolverDnSpotrs(handle, uplo, n, 1, A, lda, x, n, info));

    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: cholesky = %10.6f sec\n", time_solve);

    if (info  ) { checkCudaErrors(hipFree(info)); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }

    return 0;
}


/*
 *  solve A*x = b by LU with partial pivoting
 *
 */
int linearSolverLU(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    int bufferSize = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    int *ipiv = NULL; // pivoting sequence
    int h_info = 0;
    float start, stop;
    float time_solve;

    checkCudaErrors(hipsolverDnSgetrf_bufferSize(handle, n, n, (float*)Acopy, lda, &bufferSize));

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));
    checkCudaErrors(hipMalloc(&ipiv, sizeof(int)*n));


    // prepare a copy of A because getrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

    checkCudaErrors(hipsolverDnSgetrf(handle, n, n, A, lda, buffer, ipiv, info));
    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: LU factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, A, lda, ipiv, x, n, info));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: LU = %10.6f sec\n", time_solve);

    if (info  ) { checkCudaErrors(hipFree(info  )); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }
    if (ipiv  ) { checkCudaErrors(hipFree(ipiv));}

    return 0;
}


int linearSolverSVD(
    hipsolverHandle_t handle, 
    int n,
    const float *Acopy,
    int lda,
    const float *bcopy,
    float *x)
{
    hipblasHandle_t cublasHandle = NULL; // used in residual evaluation
    int m = lda;
    int bufferSize = 0;
    int *info = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    const float one = 1.0;

    // float U[lda*m]; // m-by-m unitary matrix 
    // float VT[lda*n]; // n-by-n unitary matrix
    // float S[n]; //singular value 
    float *d_A = NULL; float *d_SI = NULL; 
    float *d_b = NULL; float *d_S = NULL; 
    float *d_U = NULL; float *d_VT = NULL; 
    float *d_work = NULL; 
    float *d_rwork = NULL; 
    float *d_W = NULL; 
    signed char jobu = 'A'; // all m columns of U 
    signed char jobvt = 'A'; // all n columns of VT 
    // step 1: create cusolverDn/cublas handle 
    checkCudaErrors(hipblasCreate(&cublasHandle)); 

    checkCudaErrors(hipMalloc((void**)&d_A , sizeof(float)*lda*n)); \
    checkCudaErrors(hipMalloc((void**)&d_b , sizeof(float)*m)); 
    checkCudaErrors(hipMalloc((void**)&d_S , sizeof(float)*n)); 
    checkCudaErrors(hipMalloc((void**)&d_SI , sizeof(float)*lda*n)); 
    checkCudaErrors(hipMalloc((void**)&d_U , sizeof(float)*lda*m)); 
    checkCudaErrors(hipMalloc((void**)&d_VT , sizeof(float)*lda*n)); 
    checkCudaErrors(hipMalloc((void**)&info, sizeof(int))); 
    checkCudaErrors(hipMalloc((void**)&d_W , sizeof(float)*lda*n));
    checkCudaErrors(hipMemcpy(d_A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice)); //gesvd destroys d_A on exit
    checkCudaErrors(hipMemcpy(d_b, bcopy, sizeof(float)*m, hipMemcpyDeviceToDevice));
    
    // checkMatrix(m, n, d_SI, lda, "zero_SI");
    // checkMatrix(m, n , d_A, lda, "SVD_AtA");
    // checkArray(d_b, m, "SVD_Atb");
    checkCudaErrors(hipsolverDnSgesvd_bufferSize( handle, m, n, &bufferSize ));
    checkCudaErrors(hipMalloc((void**)&d_work , sizeof(float)*bufferSize));

    start = second();

    checkCudaErrors(hipsolverDnSgesvd( 
        handle, jobu, jobvt, m, n, d_A, lda, d_S, d_U, lda, d_VT, lda, d_work, bufferSize, d_rwork, info));
    //checkCudaErrors(hipDeviceSynchronize());
    
    // checkCudaErrors(hipMemcpy(U , d_U , sizeof(float)*lda*m, hipMemcpyDeviceToHost)); 
    // checkCudaErrors(hipMemcpy(VT, d_VT, sizeof(float)*lda*n, hipMemcpyDeviceToHost)); 
    // checkCudaErrors(hipMemcpy(S , d_S , sizeof(float)*n , hipMemcpyDeviceToHost)); 
    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: SVD failed, check %d parameter\n", h_info);
    }

    // int BLOCK_DIM_X = 32; int BLOCK_DIM_Y = 32;
    // dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);  
    // dim3 gridDim((n + BLOCK_DIM_X - 1) / BLOCK_DIM_X, (m + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);
    // initSIGPU<<<gridDim, blockDim>>>(d_SI, d_S, m, n);
    float epsilon = 1.e-9;
    printf("epsilon = %f \n", epsilon);
    initSI<float>(d_SI, d_S, m, n, epsilon, 256);
    //int initStat = initSICPU(d_SI, d_S, m, n, epsilon);
    // U*S*V*x=b; x = VT*Si*UT*b
    // checkMatrix(m, n, d_SI, lda, "SVD_SI");
    // checkArray(d_S, n, "dS");
    // checkMatrix(m, m, d_U, lda, "SVD_U");
    // checkMatrix(n, n, d_VT, lda, "SVD_VT");
    float al = 1.0;// al =1
    float bet = 0.0;// bet =0
    // checkArray(d_b, n, "db");
    checkCudaErrors(hipblasSgemv(cublasHandle,HIPBLAS_OP_T, m, m, &al,d_U, m, d_b,1,&bet,d_b,1));
    // checkArray(d_b, n, "dUtb");
    checkCudaErrors(hipblasSgemv(cublasHandle,HIPBLAS_OP_N, m, n, &al,d_SI, m, d_b,1,&bet,d_b,1));
    // checkArray(d_b, n, "dSiUtb");
    checkCudaErrors(hipblasSgemv(cublasHandle,HIPBLAS_OP_T, n, n, &al,d_VT, n, d_b, 1,&bet,x,1));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_solve = stop - start; 
    fprintf (stdout, "timing: SVD = %10.6f sec\n", time_solve);
    // checkArray(x, 20, "d_x");

    if (d_A ) hipFree(d_A); 
    if (d_S ) hipFree(d_S); 
    if (d_SI ) hipFree(d_SI);
    if (d_U ) hipFree(d_U); 
    if (d_VT ) hipFree(d_VT); 
    if (info) hipFree(info); 
    if (d_work ) hipFree(d_work); 
    if (d_rwork) hipFree(d_rwork); 
    if (d_W ) hipFree(d_W); 
    if (cublasHandle ) hipblasDestroy(cublasHandle); 
    // if (cusolverH) hipsolverDnDestroy(cusolverH); 
    return 0;

}

/*
 *  solve A*x = b by QR
 *
 */
int linearSolverQR(
    hipsolverHandle_t handle,
    int n,
    const float *Acopy,
    int lda,
    const float *b,
    float *x)
{
    hipblasHandle_t cublasHandle = NULL; // used in residual evaluation
    int bufferSize = 0;
    int bufferSize_geqrf = 0;
    int bufferSize_ormqr = 0;
    int *info = NULL;
    float *buffer = NULL;
    float *A = NULL;
    float *tau = NULL;
    int h_info = 0;
    float start, stop;
    float time_solve;
    const float one = 1.0;

    checkCudaErrors(hipblasCreate(&cublasHandle));

    checkCudaErrors(hipsolverDnSgeqrf_bufferSize(handle, n, n, (float*)Acopy, lda, &bufferSize_geqrf));
    checkCudaErrors(hipsolverDnSormqr_bufferSize(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        n,
        1,
        n,
        A,
        lda,
        NULL,
        x,
        n,
        &bufferSize_ormqr));

    //printf("buffer_geqrf = %d, buffer_ormqr = %d \n", bufferSize_geqrf, bufferSize_ormqr);
    
    bufferSize = (bufferSize_geqrf > bufferSize_ormqr)? bufferSize_geqrf : bufferSize_ormqr ; 

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(float)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(float)*lda*n));
    checkCudaErrors(hipMalloc ((void**)&tau, sizeof(float)*n));

// prepare a copy of A because getrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(float)*lda*n, hipMemcpyDeviceToDevice));

    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    start = second();
    start = second();

// compute QR factorization
    checkCudaErrors(hipsolverDnSgeqrf(handle, n, n, A, lda, tau, buffer, bufferSize, info));

    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: QR factorization failed, check %d parameter\n", h_info);
    }

    checkCudaErrors(hipMemcpy(x, b, sizeof(float)*n, hipMemcpyDeviceToDevice));

    // compute Q^T*b
    checkCudaErrors(hipsolverDnSormqr(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        n,
        1,
        n,
        A,
        lda,
        tau,
        x,
        n,
        buffer,
        bufferSize,
        info));

    // x = R \ Q^T*b
    checkCudaErrors(hipblasStrsm(
         cublasHandle,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N,
         HIPBLAS_DIAG_NON_UNIT,
         n,
         1,
         &one,
         A,
         lda,
         x,
         n));
    checkCudaErrors(hipDeviceSynchronize());
    stop = second();

    time_solve = stop - start;
    fprintf (stdout, "timing: QR = %10.6f sec\n", time_solve);

    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    if (info  ) { checkCudaErrors(hipFree(info  )); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }
    if (tau   ) { checkCudaErrors(hipFree(tau)); }

    return 0;
}


DnSolver::DnSolver (int rows_, int cols_) 
{
    DnSolver::~DnSolver();

    rowsA = rows_;
    colsA = cols_;
    lda = rows_;

    checkCudaErrors(hipsolverDnCreate(&handle));
    checkCudaErrors(hipblasCreate(&cublasHandle));
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipsparseCreate(&cusparseHandle));
    checkCudaErrors(hipsparseCreateMatDescr(&descrA));

    checkCudaErrors(hipsolverSetStream(handle, stream));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream));


    h_A = (float*)malloc(sizeof(float)*lda*colsA);
    h_x = (float*)malloc(sizeof(float)*colsA);
    h_b = (float*)malloc(sizeof(float)*rowsA);

    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(float)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(float)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_A, sizeof(float)*lda*colsA)); 


}

void DnSolver::from_dense(float* array_host_, float* rhs_){

    h_A = array_host_;
    h_b = rhs_;
    checkCudaErrors(hipMemcpy(d_A, h_A, sizeof(float)*lda*colsA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, sizeof(float)*rowsA, hipMemcpyHostToDevice));
}


void DnSolver::from_csr(int* indptr_, int* indices_, float* data_, float* rhs_){
    
    h_b = rhs_;
    h_csrRowPtrA = indptr_;
    h_csrColIndA = indices_;
    h_csrValA = data_;
    baseA = h_csrRowPtrA[0];
    nnzA = h_csrRowPtrA[rowsA] - baseA;
    //checkMatrix(nnzA, 1, h_csrValA, nnzA, "h_valA");
    hipsparseStatus_t cpstat;

    hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); 
    hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);
    if (d_csrRowPtrA == NULL ){
        printf("allocating pointers \n");
        checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, sizeof(int)*(rowsA+1)));
        checkCudaErrors(hipMalloc((void **)&d_csrColIndA, sizeof(int)*nnzA));
        checkCudaErrors(hipMalloc((void **)&d_csrValA, sizeof(float)*nnzA));
    }
    checkCudaErrors(hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, sizeof(int)*(rowsA+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIndA, h_csrColIndA, sizeof(int)*nnzA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA, h_csrValA, sizeof(float)*nnzA, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, sizeof(float)*rowsA, hipMemcpyHostToDevice));

    cpstat = hipsparseScsr2dense(
                    cusparseHandle, 
                    rowsA, colsA, 
                    descrA, 
                    d_csrValA, 
                    d_csrRowPtrA,
                    d_csrColIndA, 
                    d_A, rowsA);
    if (cpstat != HIPSPARSE_STATUS_SUCCESS) { 
        printf ("%s\n", "CuSparse CSR to dense conversion failed"); 
        return; 
    } 

    //if (d_csrValA   ) { checkCudaErrors(hipFree(d_csrValA)); }
    //if (d_csrRowPtrA) { checkCudaErrors(hipFree(d_csrRowPtrA)); }
    //if (d_csrColIndA) { checkCudaErrors(hipFree(d_csrColIndA)); }

    //cpstat = hipsparseScsrmm(cusparseHandle,
    //                        HIPSPARSE_OPERATION_TRANSPOSE,
    //                        colsA,colsA,rowsA,nnzA, &al,
    //                        descrA, 
    //                        d_csrValA,
    //                        h_csrRowPtrA, 
    //                        d_csrColIndA, rowsA,d_A,rowsA,&bet,dAtA,colsA);


}

void DnSolver::solve(int Func) {
    //printf("step 6: compute AtA \n");
    hipblasStatus_t cbstat;
    float* dAtA;
    checkCudaErrors(hipMalloc(&dAtA, sizeof(float)*colsA*colsA));

    cbstat = hipblasSgemm(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_N,colsA,colsA,rowsA,&al,d_A,rowsA,d_A,rowsA,&bet,dAtA,colsA);

    // printf("step 7: compute At*b \n");
    float* d_Atb;
    checkCudaErrors(hipMalloc((void **)&d_Atb, sizeof(float)*colsA));
    cbstat = hipblasSgemv(cublasHandle,HIPBLAS_OP_T,rowsA,colsA,&al,d_A,rowsA,d_b,1,&bet,d_Atb,1);

    //print out for debug
    //checkMatrix(rowsA, colsA , d_A, lda, "A");
    //checkMatrix(rowsA, 1 , d_b, rowsA, "b");
    //checkMatrix(colsA, colsA , dAtA, lda, "AtA");
    //checkMatrix(colsA, 1 , d_Atb, lda, "Atb");

    //if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    //checkCudaErrors(hipblasCreate(&cublasHandle));
    //checkCudaErrors(hipblasSetStream(cublasHandle, stream));
    //printf("step 8: solves AtA*x = At*b \n");

    if ( 0 == Func )
    {
        linearSolverQR(handle, colsA, dAtA, colsA, d_Atb, d_x);
    }
    else if ( 1 == Func )
    {
        linearSolverCHOL(handle, colsA, dAtA, colsA, d_Atb, d_x);
    }
    else if ( 2 == Func )
    {
        linearSolverLU(handle, colsA, dAtA, colsA, d_Atb, d_x);
    }
    else if ( 3 == Func )
    {
        linearSolverSVD(handle, colsA, dAtA, colsA, d_Atb, d_x);
    }
    else
    {
        fprintf(stderr, "Error: %d is unknown function\n", Func);
        exit(EXIT_FAILURE);
    }
    if (dAtA) { checkCudaErrors(hipFree(dAtA)); }
    if (d_Atb) { checkCudaErrors(hipFree(d_Atb)); }
    //if (d_A) { checkCudaErrors(hipFree(d_A)); }
    //if (d_b) { checkCudaErrors(hipFree(d_b)); }

}
void DnSolver::solve_Axb(int Func) {


    if ( 0 == Func )
    {
        linearSolverQR(handle, colsA, d_A, colsA, d_b, d_x);
    }
    else if ( 1 == Func )
    {
        linearSolverCHOL(handle, colsA, d_A, colsA, d_b, d_x);
    }
    else if ( 2 == Func )
    {
        linearSolverLU(handle, colsA, d_A, colsA, d_b, d_x);
    }
    else if ( 3 == Func )
    {
        linearSolverSVD(handle, colsA, d_A, colsA, d_b, d_x);
    }
    else
    {
        fprintf(stderr, "Error: %d is unknown function\n", Func);
        exit(EXIT_FAILURE);
    }
    //if (d_A) { checkCudaErrors(hipFree(d_A)); }
    //if (d_b) { checkCudaErrors(hipFree(d_b)); }

}
void DnSolver::retrieve_to(float* h_x)
{
    checkCudaErrors(hipMemcpy(h_x, d_x, sizeof(float)*colsA, hipMemcpyDeviceToHost));
    //checkCudaErrors(hipFree(d_x));
    //if (d_A) { checkCudaErrors(hipFree(d_A)); }
    //if (d_b) { checkCudaErrors(hipFree(d_b)); }
}

DnSolver::~DnSolver()
{
    if (handle) { checkCudaErrors(hipsolverDnDestroy(handle)); }
    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    if (cusparseHandle) { checkCudaErrors(hipsparseDestroy(cusparseHandle)); }
    if (descrA) { checkCudaErrors(hipsparseDestroyMatDescr(descrA)); }
    if (stream) { checkCudaErrors(hipStreamDestroy(stream)); }

    if (h_A) { free(h_A); }
    if (h_x) { free(h_x); }
    if (h_b) { free(h_b); }

    if (h_csrValA   ) { free(h_csrValA); }
    if (h_csrRowPtrA) { free(h_csrRowPtrA); }
    if (h_csrColIndA) { free(h_csrColIndA); }

    if (d_A) { checkCudaErrors(hipFree(d_A)); }
    if (d_x) { checkCudaErrors(hipFree(d_x)); }
    if (d_b) { checkCudaErrors(hipFree(d_b)); }


    if (d_csrValA   ) { checkCudaErrors(hipFree(d_csrValA)); }
    if (d_csrRowPtrA) { checkCudaErrors(hipFree(d_csrRowPtrA)); }
    if (d_csrColIndA) { checkCudaErrors(hipFree(d_csrColIndA)); }
}
