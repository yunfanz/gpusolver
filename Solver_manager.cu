#include "Solver_manager.hh"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <cmath> 
#include <hip/hip_runtime.h>
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"
#include "helper_cusolver.h"
#include <chrono>
#include <iostream>
#include <complex>
#include <hip/hip_complex.h>
#include "corrMatrix.cu"



DnSolver::DnSolver (int rows_) 
{
    DnSolver::~DnSolver();

    rowsA = rows_;
    colsA = rows_;
    lda = rows_;

    checkCudaErrors(hipsolverDnCreate(&cusolverH));
    checkCudaErrors(hipblasCreate(&cublasHandle));
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipsolverSetStream(cusolverH, stream));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream));

    h_A = (std::complex<float>*)malloc(sizeof(std::complex<float>)*lda*colsA);
    h_V = (std::complex<float>*)malloc(sizeof(std::complex<float>)*rowsA);
    h_S = (float*)malloc(sizeof(float)*colsA);

    checkCudaErrors(hipMalloc((void **)&d_S, sizeof(float)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_V, sizeof(hipComplex)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_A, sizeof(hipComplex)*lda*colsA)); 

    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int))); 

    checkCudaErrors(hipsolverDnCheevd_bufferSize( cusolverH, jobz, uplo, lda, d_A, lda, d_S, &lwork));

    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipComplex)*lwork));


}



void DnSolver::corr_from_vec(std::complex<float>* vec_host_){

    h_V = vec_host_;
    checkCudaErrors(hipMemcpy(d_V, h_V, sizeof(hipComplex)*lda, hipMemcpyHostToDevice));
    get_correlation(d_A, d_V, lda, 256);
    //checkMatrix(lda, 1, d_V, lda, "dV");
    //checkMatrix(lda, lda, d_A, lda, "dA");

}



void DnSolver::solve() {
    
    int n = lda;
    int h_info = 0;
    float start, stop;
    float time_solve;

    start = second();

    checkCudaErrors(hipsolverDnCheevd( cusolverH, jobz, uplo, n, d_A, lda, d_S, d_work, lwork, devInfo));
    //checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipMemcpy(&h_info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

    if ( 0 != h_info ){
        fprintf(stderr, "Error: EVD failed, check %d parameter\n", h_info);
    }

   
    stop = second();
    time_solve = stop - start; 
    //fprintf (stdout, "timing: EVD = %10.6f sec\n", time_solve);

    //if (d_A ) hipFree(d_A); 
    //if (d_S ) hipFree(d_S); 
    //if (devInfo) hipFree(devInfo); 
    //if (d_work ) hipFree(d_work); 
    //if (cublasHandle ) hipblasDestroy(cublasHandle); 
    //if (cusolverH) hipsolverDnDestroy(cusolverH); 
}


void DnSolver::retrieve_to(float* h_S)
{
    checkCudaErrors(hipMemcpy(h_S, d_S, sizeof(float)*colsA, hipMemcpyDeviceToHost));
}

DnSolver::~DnSolver()
{
    if (cusolverH) { checkCudaErrors(hipsolverDnDestroy(cusolverH)); }
    if (cublasHandle) { checkCudaErrors(hipblasDestroy(cublasHandle)); }
    if (stream) { checkCudaErrors(hipStreamDestroy(stream)); }

    if (h_A) { free(h_A); }
    if (h_S) { free(h_S); }
    if (h_V) { free(h_V); }


    if (d_A) { checkCudaErrors(hipFree(d_A)); }
    if (d_S) { checkCudaErrors(hipFree(d_S)); }
    if (d_V) { checkCudaErrors(hipFree(d_V)); }

}
